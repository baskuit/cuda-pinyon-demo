#include "hip/hip_runtime.h"
// #include <stdio.h>

#include "./common.hh"

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

void alloc_buffers (
    float **host_ptr, 
    float** device_ptr, 
    const long int batch_size, 
    const long int input_size) {

}

void cuda_test()
{
    int N = 1 << 20;
    float *x, *y, *d_x, *d_y;
    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
}
