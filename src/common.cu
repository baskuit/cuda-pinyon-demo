#include "hip/hip_runtime.h"
// #include <stdio.h>

#include <math.h>

#include "./common.hh"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ hiprandState *devStates;

__global__ void __convert_kernel(const uint64_t *input, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 47 * 8)
    {
        int input_index = tid / 8;
        int byte_index = tid % 8;
        uint64_t value = input[input_index];
        value >>= (64 - (byte_index * 8));
        value &= 0xFF;
        output[tid] = (float)value;
    }
}

__device__ float __generate(hiprandState* globalState, int ind)
{
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void __sample_kernel(
    BufferData tgt,
    BufferData src,
    const int block_size,
    const int start_block_index)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    const int block_index = start_block_index + blockIdx.x;
    const int base_sample_index = block_index * block_size;

    // int sample_index = (int)(ceil((hiprand_uniform(&devStates) * (blockDim.x + 1))) - 1);
    float r = __generate(devStates, blockIdx.x);

    // output[tid] = (float)value;
}

__global__ void __setup_kernel ( unsigned long seed )
{
    hipMalloc(&devStates, gridDim.x * sizeof(hiprandState));
    hiprand_init ( seed, blockIdx.x, 0, &devStates[blockIdx.x] );
}




void sample(
    BufferData tgt,
    BufferData src,
    const int block_size,
    const int start_block_index,
    const int num_blocks_to_sample,
    const int num_samples_per_block)
{
    dim3 gridDim(num_blocks_to_sample, 1, 1);
    dim3 blockDim(num_samples_per_block, 1, 1);
    __sample_kernel<<<gridDim, blockDim>>>(tgt, src, block_size, start_block_index);
};

void convert(
    float *output,
    const uint64_t *input)
{
    dim3 gridDim(12, 1, 1);
    dim3 blockDim(32, 1, 1);
    __convert_kernel<<<gridDim, blockDim>>>(input, output);
}

void copy(
    uint64_t *dest,
    const uint64_t *src,
    const int len)
{
    hipMemcpy(dest, src, len, hipMemcpyHostToDevice);
}

void copy_battle(
    uint64_t *raw_bytes,
    const uint64_t *battle_bytes,
    const int index)
{
    hipMemcpy(raw_bytes, battle_bytes, 376, hipMemcpyHostToDevice);
}

void alloc_buffers(
    uint64_t **raw_buffer,
    float **float_buffer,
    // float **joined_policy_buffer,
    const long int batch_size)
{

    hipHostMalloc(raw_buffer, batch_size * 47 * sizeof(uint64_t));
    hipHostMalloc(float_buffer, batch_size * 376 * sizeof(float));
    // hipHostMalloc(joined_policy_buffer, batch_size * 18 * sizeof(float));
}

void alloc_buffers2(
    uint64_t **raw_buffer,
    float **float_buffer,
    // float **joined_policy_buffer,
    const long int batch_size)
{

    hipHostMalloc(raw_buffer, batch_size * 47 * sizeof(uint64_t));
    hipHostMalloc(float_buffer, batch_size * 376 * sizeof(float));
    //
}

void dealloc_buffers(
    uint64_t **raw_buffer,
    float **float_buffer)
{
    hipFree(raw_buffer);
    hipFree(float_buffer);
}

void setup_rng(const int n_blocks)
{
    srand(time(0));
    int seed = rand();
    __setup_kernel<<<n_blocks, 1>>>(seed);
}
