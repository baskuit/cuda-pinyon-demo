#include "hip/hip_runtime.h"
// #include <stdio.h>

#include <math.h>

#include "./common.hh"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

hiprandState *devStates;

__global__ void __convert_kernel(const uint64_t *input, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 47 * 8)
    {
        int input_index = tid / 8;
        int byte_index = tid % 8;
        uint64_t value = input[input_index];
        value >>= (64 - (byte_index * 8));
        value &= 0xFF;
        output[tid] = (float)value;
    }
}

__global__ void __sample_kernel(
    BufferData tgt,
    BufferData src,
    const int start_block_index,
    const int end_block_index)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // int sample_index = (int)(ceil((hiprand_uniform(&state) * (blockDim.x + 1))) - 1);

    // output[tid] = (float)value;
}

__global__ void __setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
}

__device__ float generate(hiprandState* globalState, int ind)
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}


void sample(
    BufferData tgt,
    BufferData src,
    const int start_block_index,
    const int end_block_index,
    const int num_samples)
{
    dim3 gridDim(end_block_index - start_block_index, 1, 1);
    dim3 blockDim(num_samples, 1, 1);
    __sample_kernel<<<gridDim, blockDim>>>(tgt, src, start_block_index, end_block_index);
};

void convert(
    float *output,
    const uint64_t *input)
{
    dim3 gridDim(12, 1, 1);
    dim3 blockDim(32, 1, 1);
    __convert_kernel<<<gridDim, blockDim>>>(input, output);
}

void copy(
    uint64_t *dest,
    const uint64_t *src,
    const int len)
{
    hipMemcpy(dest, src, len, hipMemcpyHostToDevice);
}

void copy_battle(
    uint64_t *raw_bytes,
    const uint64_t *battle_bytes,
    const int index)
{
    hipMemcpy(raw_bytes, battle_bytes, 376, hipMemcpyHostToDevice);
}

void alloc_buffers(
    uint64_t **raw_buffer,
    float **float_buffer,
    // float **joined_policy_buffer,
    const long int batch_size)
{

    hipHostMalloc(raw_buffer, batch_size * 47 * sizeof(uint64_t));
    hipHostMalloc(float_buffer, batch_size * 376 * sizeof(float));
    // hipHostMalloc(joined_policy_buffer, batch_size * 18 * sizeof(float));
}

void alloc_buffers2(
    uint64_t **raw_buffer,
    float **float_buffer,
    // float **joined_policy_buffer,
    const long int batch_size)
{

    hipHostMalloc(raw_buffer, batch_size * 47 * sizeof(uint64_t));
    hipHostMalloc(float_buffer, batch_size * 376 * sizeof(float));
    //
}

void dealloc_buffers(
    uint64_t **raw_buffer,
    float **float_buffer)
{
    hipFree(raw_buffer);
    hipFree(float_buffer);
}

void setup_rng(const int n_threads)
{
    hipMalloc(&devStates, n_threads * sizeof(hiprandState));
    srand(time(0));
    int seed = rand();
    __setup_kernel<<<2, 5>>>(devStates, seed);
}
